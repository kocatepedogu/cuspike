// SPDX-FileCopyrightText: 2025 Doğu Kocatepe
// SPDX-License-Identifier: GPL-3.0-or-later

#include <stdint.h>

#include "config.hpp"
#include "util.hpp"
#include "initialize.hpp"
#include "kernel-register.hpp"
#include "kernel-global.hpp"
#include "save.hpp"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>

int numBlocksPerSmForRegisterKernel;
int numBlocksPerSmForGlobalKernel;
int numBlocksForRegisterKernel;
int numBlocksForGlobalKernel;

uint32_t *spike_times = nullptr;
uint32_t *spike_counts = nullptr;

uint32_t *matrix = nullptr;
uint32_t *synapses = nullptr;
uint32_t *indices = nullptr;

hipStream_t createStreamWithResidency(int dev, void *ptr) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, dev);
    CUDA_CALL(hipDeviceSetLimit(cudaLimitPersistingL2CacheSize, prop.persistingL2CacheMaxSize));

    hipStream_t stream;
    CUDA_CALL(hipStreamCreate(&stream));

    hipLaunchAttributeValue stream_attribute;
    stream_attribute.accessPolicyWindow.base_ptr  = reinterpret_cast<void*>(ptr);
    stream_attribute.accessPolicyWindow.num_bytes = prop.persistingL2CacheMaxSize;
    stream_attribute.accessPolicyWindow.hitRatio  = 1.0;
    stream_attribute.accessPolicyWindow.hitProp   = hipAccessPropertyPersisting;
    stream_attribute.accessPolicyWindow.missProp  = hipAccessPropertyStreaming;
    CUDA_CALL(hipStreamSetAttribute(stream, hipLaunchAttributeAccessPolicyWindow, &stream_attribute));

    return stream;
}

void launchRegisterKernel(int dev) {
    dim3 dimBlock(numThreads, 1, 1);
    dim3 dimGrid(numBlocksForRegisterKernel, 1, 1);

    CUDA_CALL(hipMallocManaged(&matrix, 1 + N * N * sizeof *matrix / 32));
    initialize_synapses_bitmap(dev);

    void *kernelArgs[] = {&spike_times, &spike_counts, &matrix};

    if constexpr (l2residency) {
        hipStream_t stream = createStreamWithResidency(dev, matrix);
        CUDA_CALL(hipLaunchCooperativeKernel((void*)simulate_register, dimGrid, dimBlock, kernelArgs, 0, stream));
    }
    else {
        CUDA_CALL(hipLaunchCooperativeKernel((void*)simulate_register, dimGrid, dimBlock, kernelArgs));
    }
}

void launchGlobalKernel(int dev) {
    dim3 dimBlock(numThreads, 1, 1);
    dim3 dimGrid(numBlocksForGlobalKernel, 1, 1);

    CUDA_CALL(hipMallocManaged(&synapses, N * N * sizeof *synapses));
    CUDA_CALL(hipMallocManaged(&indices, (N + 1) * sizeof *indices));
    initialize_synapses_csr(dev);

    void *kernelArgs[] = {&spike_times, &spike_counts, &synapses, &indices};

    if constexpr(l2residency) {
        hipStream_t stream = createStreamWithResidency(dev, synapses);
        CUDA_CALL(hipLaunchCooperativeKernel((void*)simulate_global, dimGrid, dimBlock, kernelArgs, 0, stream));
    }
    else {
        CUDA_CALL(hipLaunchCooperativeKernel((void*)simulate_global, dimGrid, dimBlock, kernelArgs));
    }
}

int main(int argc, char **argv) {
    int dev = 0;
    int supportsCoopLaunch = 0;

    CUDA_CALL(hipDeviceGetAttribute(&supportsCoopLaunch, hipDeviceAttributeCooperativeLaunch, dev));
    if( supportsCoopLaunch != 1 ) {
        printf("Cooperative Launch is not supported on this machine.\n");
        abort();
    }

    hipDeviceProp_t deviceProp;
    CUDA_CALL(hipGetDeviceProperties(&deviceProp, dev));

    CUDA_CALL(hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSmForRegisterKernel, simulate_register, numThreads, 0));
    CUDA_CALL(hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSmForGlobalKernel, simulate_global, numThreads, 0));

    numBlocksForRegisterKernel = numBlocksPerSmForRegisterKernel * deviceProp.multiProcessorCount;
    numBlocksForGlobalKernel = numBlocksPerSmForGlobalKernel * deviceProp.multiProcessorCount;

    CUDA_CALL(hipMallocManaged(&spike_times, N * steps * sizeof *spike_times));
    CUDA_CALL(hipMallocManaged(&spike_counts, N * sizeof *spike_counts));

    if (N < numBlocksForRegisterKernel * numThreads) {
        printf("Using registers to store state variables.\n");
        launchRegisterKernel(dev);
    } else {
        printf("Using global memory to store state variables.\n");
        launchGlobalKernel(dev);
    }

    CUDA_CALL(hipDeviceSynchronize());

    if (argc == 2 && !strcmp(argv[1], "plot")) {
        print_results();
    }

    CUDA_CALL(hipFree(spike_times));
    CUDA_CALL(hipFree(spike_counts));

    return 0;
}
